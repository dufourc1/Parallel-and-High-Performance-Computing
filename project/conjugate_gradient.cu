#include "hip/hip_runtime.h"
#include "cg.hh"
#include "matrix.hh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cblas.h>

const double NEARZERO = 1.0e-14;
const bool DEBUG = true;

// ouput = A*x
// only work for one dimensional grid: each row is processed by one block at most
__global__ void matrix_vector(double *A, double *x, double *output, int n)
{
    // stupid implementation where each thread computes one element of the output
    extern __shared__ double row_sums[];
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x * blockDim.y + threadIdx.y;
    if (row < n)
    {
        double sum = 0;
        // blockIdx.y should be 1 for this kernel to work
        for (int j = threadIdx.y; j < n; j += blockDim.y)
        {
            sum += A[row * n + j] * x[j];
        }

        // store result in shared memory
        row_sums[tid] = sum;

        // wait for all threads in the block to finish and then aggregate the results
        __syncthreads();
        // jmp >>= 1 is equivalent to jmp /= 2
        for (int jmp = blockDim.y / 2; jmp > 0; jmp >>= 1)
        {
            // first iteration, each thread in the first half of the block adds the result of the second half to its result
            if (threadIdx.y < jmp)
            {
                row_sums[tid] += row_sums[tid + jmp];
            }
            __syncthreads();
        }

        // first thread in the row writes the result to global memory since it aggregated all the results
        if (threadIdx.y == 0)
        {
            output[row] = row_sums[tid];
        }
    }
}

// output = x  + y * scale_y
__global__ void scale_add_vector(double *x, double *y, double *scale_y, double *output, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
    {
        output[index] = x[index] + y[index] * *scale_y;
    }
}

// output = x  - y * scale_y
__global__ void scale_subtract_vector(double *x, double *y, double *scale_y, double *output, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
    {
        output[index] = x[index] - y[index] * *scale_y;
    }
}

__global__ void diff_vector(double *x, double *y, double *output, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
    {
        output[index] = x[index] - y[index];
    }
}

// output = copy(scale)
__global__ void copy_vector(double *x, double *output, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
    {
        output[index] = x[index];
    }
}

// output = x/y
__global__ void div_scalar(double *x, double *y, double *output)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0)
    {
        *output = *x / max(*y * NEARZERO, *y);
    }
}

// y = copy(x)
__global__ void copy_scalar(double *x, double *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0)
    {
        *y = *x;
    }
}

void print_gpu_value(double *x, int n)
{
    double *x_host = new double[n];
    hipMemcpy(x_host, x, n * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++)
    {
        std::cout << x_host[i] << " ";
    }
    std::cout << std::endl;
    delete[] x_host;
}

void CGSolver::solve_CUDA(double *A, double *b, double *x)
{
    double r_norm;

    // device memory allocation for vectors
    double *r;
    double *p;
    double *temp;

    hipMalloc((void **)&r, m_m * sizeof(double));
    hipMalloc((void **)&p, m_n * sizeof(double));
    hipMalloc((void **)&temp, m_m * sizeof(double));

    // device memory allocation for scalars
    double *rsold;
    double *rsnew;
    double *alpha;
    double *beta;
    double *scalar_temp;

    hipMalloc((void **)&rsold, sizeof(double));
    hipMalloc((void **)&rsnew, sizeof(double));
    hipMalloc((void **)&alpha, sizeof(double));
    hipMalloc((void **)&beta, sizeof(double));
    hipMalloc((void **)&scalar_temp, sizeof(double));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    int rows_per_block = 32;
    int threads_per_line = 32;
    int shared_mem = rows_per_block * threads_per_line * sizeof(double);

    dim3 matrix_block_size(rows_per_block, threads_per_line);
    // one row is assigned to one block (not more) →  only a one dimensional grid
    dim3 matrix_grid_size((m_n + matrix_block_size.x - 1) / matrix_block_size.x);

    // 1 thread per index
    dim3 vector_block_size((m_m + 31) / 32);
    dim3 vector_grid_size((m_n + vector_block_size.x - 1) / vector_block_size.x);

    // r = b - A*x
    matrix_vector<<<matrix_grid_size, matrix_block_size, shared_mem>>>(A, x, temp, m_n);
    diff_vector<<<vector_grid_size, vector_block_size>>>(b, temp, r, m_n);
    copy_vector<<<vector_grid_size, vector_block_size>>>(r, p, m_n);

    hipblasDdot(handle, m_n, p, 1, p, 1, rsold);

    /*
    We don't need hipDeviceSynchronize() here because all the kernels launched by the same stream are executed sequentially.
    */

    int k = 0;
    if (max_iter == -1)
    {
        max_iter = m_n;
    }
    for (; k < max_iter; ++k)
    {
        // temp = A*p
        matrix_vector<<<matrix_grid_size, matrix_block_size, shared_mem>>>(A, p, temp, m_n);

        // alpha = rsold / (p^T temp)
        hipblasDdot(handle, m_n, p, 1, temp, 1, scalar_temp);
        div_scalar<<<1, 1>>>(rsold, scalar_temp, alpha);

        // x = x + alpha*p
        scale_add_vector<<<vector_grid_size, vector_block_size>>>(x, p, alpha, x, m_n);

        // r = r - alpha*Ap
        scale_subtract_vector<<<vector_grid_size, vector_block_size>>>(r, temp, alpha, r, m_n);

        // rsnew = r^T r
        hipblasDdot(handle, m_n, r, 1, r, 1, rsnew);

        // check convergence
        hipMemcpy(&r_norm, rsnew, sizeof(double), hipMemcpyDeviceToHost);

        if (DEBUG && k % 100 == 0)
        {
            std::cout << "\t[STEP " << k << "] residual = " << std::scientific
                      << std::sqrt(r_norm) << "\r" << std::endl;
        }
        if (std::sqrt(r_norm) < m_tolerance)
            break;

        // beta = rsnew / rsold
        div_scalar<<<1, 1>>>(rsnew, rsold, beta);

        // p = r + beta*p
        scale_add_vector<<<vector_grid_size, vector_block_size>>>(r, p, beta, p, m_n);

        // rsold = rsnew
        copy_scalar<<<1, 1>>>(rsnew, rsold);
    }
    if (DEBUG)
    {
        std::cout << "Converged in " << k << " iterations. Residual " << std::scientific << std::sqrt(r_norm) << std::endl;
        std::cout << "norm double sqrt =" << std::sqrt(std::sqrt(r_norm)) << std::endl;
    }
    hipDeviceSynchronize();
    hipblasDestroy(handle);

    hipFree(r);
    hipFree(p);
    hipFree(temp);

    hipFree(rsold);
    hipFree(rsnew);
    hipFree(alpha);
    hipFree(beta);
    hipFree(scalar_temp);
}

/*----------------------------------------*/
